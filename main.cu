#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "c_utils.h"
#include "hash_utils.h"
#include "md5_kernel.h"
#include "md5.h"
#include "cuda_utils.cu"

int main() {
    ERROR_CHECK(hipSetDevice(0));
    ERROR_CHECK(hipSetDeviceFlags(hipDeviceScheduleSpin));

    unsigned int content_len;
    unsigned char * file_content = (unsigned char *)malloc(sizeof(char) * FILE_MAX_LEN);
    get_file_contents("./files/samplefile.txt", file_content, &content_len);
    
    uint32_t a, b, c, d;
    md5(file_content, content_len, &a, &b, &c, &d);
    unsigned char * targetHash = aToTinyHash(a);
    printf("Welcome \n");
    printf("Target: %s\n", targetHash);

    unsigned char * fileContent; 
    unsigned char * copiedContent; 
    hipMalloc(&fileContent, sizeof(unsigned char) * FILE_MAX_LEN);
    hipMalloc(&copiedContent, sizeof(unsigned char) * (FILE_MAX_LEN + INPUT_MAX_LEN));
    hipMemcpy(fileContent, file_content, content_len, hipMemcpyHostToDevice);
    
    memset(g_cracked, 0, sizeof(unsigned char) * 5 * 11);
    memcpy(g_charset, CONST_CHARSET, sizeof(unsigned char) * CHARSET_LEN);
    memcpy(g_targetHash, targetHash, sizeof(unsigned char) * 6);
    memcpy(&g_fileContentLength, &content_len, sizeof(unsigned int));

    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceTarget), g_targetHash, sizeof(unsigned char) * 6, 0));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCharset), g_charset, sizeof(unsigned char) * CHARSET_LEN, 0));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(unsigned char) * 5 * 11, 0));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceFileContentLength), &g_fileContentLength, sizeof(unsigned int), 0));

    hipEvent_t clockBegin;
    hipEvent_t clockLast;
    hipEventCreate(&clockBegin);
    hipEventCreate(&clockLast);
    hipEventRecord(clockBegin, 0);

    int bytes = 98304;
    ERROR_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(run_tinyhash_crack), hipFuncAttributeMaxDynamicSharedMemorySize, bytes));
    
    run_tinyhash_crack<<<BLOCKS, MAX_THREADS_PER_BLOCK>>>(fileContent);
    ERROR_CHECK(hipDeviceSynchronize());


    ERROR_CHECK(hipMemcpyFromSymbol(&g_resultA, HIP_SYMBOL(g_deviceA), sizeof(uint64_t), 0, hipMemcpyDeviceToHost));
    ERROR_CHECK(hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(unsigned char) * 5 * 11, 0, hipMemcpyDeviceToHost));
    ERROR_CHECK(hipMemcpyFromSymbol(g_exampleHash, HIP_SYMBOL(g_deviceExampleHash), sizeof(unsigned char) * 6, 0, hipMemcpyDeviceToHost));
    hipFree(fileContent);

    printf("%lu\n", g_resultA);
    for (int i = 0; i < 5; i++) {
        printf("%s\n", g_cracked[i]);
    }
}