#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "file_utils.h"
#include "md5_utils.h"
#include "md5_kernel.h"

#define CONST_CHARSET "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789"

int main() {
    int devices;
    hipGetDeviceCount(&devices);
    hipSetDeviceFlags(hipDeviceScheduleSpin);

    unsigned int content_len;
    unsigned char * file_content = (unsigned char *)malloc(sizeof(char) * FILE_MAX_LEN);
    get_file_contents("./files/samplefile.txt", file_content, &content_len);
    
    unsigned char * hash = (unsigned char *)malloc(64);
    unsigned int hash_len;
    
    hash_len = md5(file_content, content_len, hash);
    print_hash(hash, hash_len);
}