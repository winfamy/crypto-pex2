// Documentation:
// Used https://github.com/iryont/md5-cracker for a basis on how to approach the problem and how to work with a CUDA device.
// Used https://opensource.apple.com/source/cvs/cvs-19/cvs/lib/md5.c for their implementation of MD5. Minor edits were made to how buffers were handled.

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "c_utils.h"
#include "hash_utils.h"
#include "task1_kernel.h"
#include "task2_kernel.h"
#include "md5.h"
#include "cuda_utils.cu"

void task1();
void task2();

int main() {
    ERROR_CHECK(hipSetDevice(0));
    ERROR_CHECK(hipSetDeviceFlags(hipDeviceScheduleSpin));

    printf("CyS 431 PEX2 – Hash Collider - by C1C Grady Phillips\n");
    task1();
    task2();
}

void task1() {
    unsigned char targetHash[16];
    char filename[256];
    printf("=== BEGIN TASK 1 ===\n");
    printf("Input filename:  ");
    scanf("%s", filename);
    printf("\n");

    unsigned int content_len;
    unsigned char * file_content = (unsigned char *)malloc(sizeof(char) * FILE_MAX_LEN);
    get_file_contents((const char *)filename, file_content, &content_len);

    md5(file_content, content_len, targetHash);

    printf("File MD5 Hash: ");
    print_hash(targetHash);
    printf("\nFile TinyHash Hash: ");
    print_tinyhash(targetHash);
    printf("\n\n");
    
    printf("[ ] Setting up CUDA global memory...\n");
    memset(g_cracked, 0, sizeof(unsigned char) * 5 * 11);
    memcpy(g_charset, CONST_CHARSET, sizeof(unsigned char) * CHARSET_LEN);
    memcpy(g_targetHash, targetHash, sizeof(unsigned char) * 16);
    printf("[+] Set up global memory!\n");

    printf("[ ] Copying CUDA device memory...\n");
    unsigned char * cracked;
    hipMalloc(&cracked, sizeof(unsigned char) * MAX_BLOCKS * 11);
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceTarget), g_targetHash, sizeof(unsigned char) * 16, 0));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCharset), g_charset, sizeof(unsigned char) * CHARSET_LEN, 0));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(unsigned char) * 5 * 11, 0));
    printf("[+] Set up device memory!\n");

    hipEvent_t clockBegin;
    hipEvent_t clockLast;
    hipEventCreate(&clockBegin);
    hipEventCreate(&clockLast);
    hipEventRecord(clockBegin, 0);

    int bytes = 98304;
    ERROR_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(run_task1_crack), hipFuncAttributeMaxDynamicSharedMemorySize, bytes));
    
    printf("\n[ ] Colliding hashes...\n");
    run_task1_crack<<<BLOCKS, MAX_THREADS_PER_BLOCK>>>();
    ERROR_CHECK(hipDeviceSynchronize());
    ERROR_CHECK(hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(unsigned char) * BLOCKS * 11, 0, hipMemcpyDeviceToHost));

    for (int i = 0; i < BLOCKS; i++) {
        printf("[+] Found collision: %s\n", g_cracked[i]);
        snprintf(filename, 255, "output/collision%d.txt", i);
        FILE * fp = fopen(filename, "w");
        if (fp) {
            fwrite(g_cracked[i], strlen((const char *) g_cracked[i]), 1, fp);
            printf("[+]\tWrote collision to file %s\n", filename);
            fclose(fp);
        } else {
            printf("[-]\tFailed to write collision to file %s.\n", filename);
        }
    }

    float milliseconds = 0;
    hipEventRecord(clockLast, 0);
    hipEventSynchronize(clockLast);
    hipEventElapsedTime(&milliseconds, clockBegin, clockLast);
    
    std::cout << "[+] Ran task 1 in " << milliseconds << " ms" << std::endl << std::endl << std::endl;
}

void task2() {
    uint32_t target = 100000;
    unsigned char targetHash[6];
    unsigned char filename[256];
    printf("=== BEGIN TASK 2 ===\n");
    printf("Input contract filename:  ");
    scanf("%s", filename);
    printf("\n");

    unsigned int content_len;
    unsigned char * file_content = (unsigned char *)malloc(sizeof(char) * FILE_MAX_LEN);
    unsigned char * firstPart = (unsigned char *)malloc(sizeof(char) * FILE_MAX_LEN);
    get_file_contents((const char *)filename, file_content, &content_len);
    sscanf((const char *) file_content, "%[A-Za-z:. $]%d", firstPart, &target);

    md5(file_content, content_len, targetHash);
    uint16_t firstPart_len = strlen((const char *) firstPart);

    printf("File MD5 Hash: ");
    print_hash(targetHash);
    printf("\nFile TinyHash Hash: ");
    print_tinyhash(targetHash);
    printf("\n\n");
    
    printf("[ ] Setting up CUDA global memory...\n");
    memcpy(&g_targetNum, &target, sizeof(uint32_t));
    memcpy(g_targetHash, targetHash, sizeof(unsigned char) * 3);
    memcpy(&g_firstPartLength, &firstPart_len, sizeof(uint16_t));
    memcpy(g_firstPartContent, firstPart, sizeof(unsigned char) * firstPart_len);
    printf("[+] Set up global memory!\n");

    printf("[ ] Copying CUDA device memory...\n");
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceTarget2), g_targetHash, sizeof(unsigned char) * 3, 0));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceTargetNum), &g_targetNum, sizeof(uint32_t), 0));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceFirstPartLength), &g_firstPartLength, sizeof(uint16_t), 0));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceFirstPartContent), g_firstPartContent, sizeof(unsigned char) * (g_firstPartLength + 1), 0));
    printf("[+] Set up device memory!\n");

    hipEvent_t clockBegin;
    hipEvent_t clockLast;
    hipEventCreate(&clockBegin);
    hipEventCreate(&clockLast);
    hipEventRecord(clockBegin, 0);
    
    printf("\n[ ] Colliding hashes...\n");
    run_task2_crack<<<MAX_BLOCKS, MAX_THREADS_PER_BLOCK_2>>>();
    ERROR_CHECK(hipDeviceSynchronize());
    ERROR_CHECK(hipMemcpyFromSymbol(&g_resultNum, HIP_SYMBOL(g_deviceNum), sizeof(uint32_t), 0, hipMemcpyDeviceToHost));
    if (g_resultNum != 0) {
        printf("[+] Found collision with value $%d\n", g_resultNum);

        FILE * fp = fopen("output/newcontract.txt", "w");
        if (fp) {
            fwrite(firstPart, firstPart_len, 1, fp);
            fprintf(fp, "%d", g_resultNum);
            fclose(fp);

            printf("[+]\tWrote contract to output/newcontract.txt\n");
        } else {
            printf("[-]\tFailed to write contract to output/newcontract.txt\n");
        }
    } else {
        printf("[-] Failed to find collision with value $%d\n", target);
    }

    float milliseconds = 0;
    hipEventRecord(clockLast, 0);
    hipEventSynchronize(clockLast);
    hipEventElapsedTime(&milliseconds, clockBegin, clockLast);
    
    std::cout << "[+] Ran task 2 in " << milliseconds << " ms" << std::endl;
}