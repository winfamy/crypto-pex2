#pragma once
#include "hip/hip_runtime.h"
#include ""
#include "consts.h"

#include <stdio.h>
#include <cmath>
#include <iostream>

#define ERROR_CHECK(X) { gpuAssert((X), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true){
  if(code != hipSuccess){
    std::cout << "Error: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
    if(abort){
      exit(code);
    }
  }
}